#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>

#include "gutil.h"

typedef struct _point
{
	int x;
	int y;
}
Point;

typedef struct _image_size
{
	int height;
	int width;
}
ImageSize;

typedef struct _image_xy_map
{
	float *xmap;
	float *ymap;
}
ImageXYMap;

typedef struct _image_weight
{
	float *blend_weight;
	float *ec_weight;
	float *total_weight;
}
ImageWeight;

typedef struct _const_data
{
	int height;
	int width;

	int warped_height;
	int warped_width;

	int corner_x;
	int corner_y;
}
ConstDataGPU;

hipError_t gCudaStatus;

#define CUDA_CHECK_CALL(fun, err_msg, return_code)					\
	gCudaStatus = fun;												\
	if(gCudaStatus != hipSuccess){									\
		fprintf(stderr, "error_code%d: %s", gCudaStatus, err_msg);	\
		return return_code;											\
	}

ConstDataGPU *const_data;
__constant__ ConstDataGPU dev_const_data[100];
ImageSize pano_size_;
ImageXYMap *dev_maps_;
ImageWeight *dev_weights_;
GPUImageData *dev_imgs_;
static int image_num_;
unsigned char *dev_pano_;

#define USE_STREAM 1
#define DST_IMAGE_CHANNEL 3

int testGPU()
{
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		return cudaStatus;
	else
		return 0;
}

int initGPU(int n)
{
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}
	image_num_ = n;
	const_data = (ConstDataGPU *)(malloc(n * sizeof(ConstDataGPU)));
	dev_maps_ = (ImageXYMap *)(malloc(n * sizeof(ImageXYMap)));
	dev_weights_ = (ImageWeight *)(malloc(n * sizeof(ImageWeight)));
	dev_imgs_ = (GPUImageData *)(malloc(n * sizeof(GPUImageData)));//dev_imgs_[0].data = 0;
	return 0;
}


int initdataCopy2GPU(C2GInitData *c2g_data, int pano_height, int pano_width)
{
	for(int i = 0; i < image_num_; i++)
	{
		const_data[i].warped_height = c2g_data[i].warped_height;
		const_data[i].warped_width = c2g_data[i].warped_width;
		const_data[i].height = c2g_data[i].height;
		const_data[i].width = c2g_data[i].width;
		const_data[i].corner_x = c2g_data[i].corner_x;
		const_data[i].corner_y = c2g_data[i].corner_y;

		int xy_map_size = c2g_data[i].warped_height * c2g_data[i].warped_width * sizeof(float);
		int img_size = c2g_data[i].height * c2g_data[i].width * 3 * sizeof(unsigned char);

		//	��xmap��ymap���Դ��Ϸ���ռ�
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_maps_[i].xmap), xy_map_size), "hipMalloc failed!\n", -2);
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_maps_[i].ymap), xy_map_size), "hipMalloc failed!\n", -2);

		//	��Ȩ�ؾ������Դ��Ϸ���ռ�
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_weights_[i].ec_weight), xy_map_size), "hipMalloc failed!\n", -2);
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_weights_[i].blend_weight), xy_map_size), "hipMalloc failed!\n", -2);
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_weights_[i].total_weight), xy_map_size), "hipMalloc failed!\n", -2);

		//	��ÿһ֡ͼ������Դ�
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_imgs_[i].data), img_size), "hipMalloc failed!\n", -2);

		//	��������
		CUDA_CHECK_CALL(hipMemcpy(dev_maps_[i].xmap, c2g_data[i].xmap, xy_map_size, hipMemcpyHostToDevice), 
			"hipMemcpy xmap failed!\n", -2);
		CUDA_CHECK_CALL(hipMemcpy(dev_maps_[i].ymap, c2g_data[i].ymap, xy_map_size, hipMemcpyHostToDevice), 
			"hipMemcpy ymap failed!\n", -2);
		CUDA_CHECK_CALL(hipMemcpy(dev_weights_[i].ec_weight, c2g_data[i].ec_weight, xy_map_size, hipMemcpyHostToDevice), 
			"hipMemcpy ec_weight failed!\n", -2);
		CUDA_CHECK_CALL(hipMemcpy(dev_weights_[i].blend_weight, c2g_data[i].blend_weight, xy_map_size, hipMemcpyHostToDevice), 
			"hipMemcpy blend_weight failed!\n", -2);
		CUDA_CHECK_CALL(hipMemcpy(dev_weights_[i].total_weight, c2g_data[i].total_weight, xy_map_size, hipMemcpyHostToDevice), 
			"hipMemcpy blend_weight failed!\n", -2);
	}
	//	�����洢��
	CUDA_CHECK_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_const_data), const_data, image_num_ * sizeof(ConstDataGPU)), 
		"hipMemcpyToSymbol failed\n", -2);

	pano_size_.height = pano_height;
	pano_size_.width = pano_width;
	int pano_malloc_size = pano_height * pano_width * DST_IMAGE_CHANNEL * sizeof(unsigned char);
	//	��ȫ��ͼ������Դ��Ϸ���ռ�
	CUDA_CHECK_CALL(hipMalloc((void**)&(dev_pano_), pano_malloc_size), "hipMalloc failed!\n", -2);
	return 0;
}

__global__ void compose(unsigned char *image, ImageXYMap xymap, ImageWeight weight, unsigned char *dst, int img_idx, ImageSize pano_size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if((i < dev_const_data[img_idx].warped_width) && (j < dev_const_data[img_idx].warped_height))
	{
		int data_idx = j * dev_const_data[img_idx].warped_width + i;
		float map_x = xymap.xmap[data_idx];
		int map_x1 = (int)map_x;
		if(map_x1 >= 0)
		{
			float map_y = xymap.ymap[data_idx];
			int map_y1 = (int)map_y;
			int map_x2 = map_x1 + 1;
			int map_y2 = map_y1 + 1;

			int dst_data_idx = ((j + dev_const_data[img_idx].corner_y) * pano_size.width + i + dev_const_data[img_idx].corner_x) * DST_IMAGE_CHANNEL;
			
			float dx1 = map_x - map_x1;
			float dy1 = map_y - map_y1;
			float dx2 = map_x2 - map_x;
			float dy2 = map_y2 - map_y;
			int img_data_idx11 = (map_y1 * dev_const_data[img_idx].width + map_x1) * 3;
			int img_data_idx12 = (map_y2 * dev_const_data[img_idx].width + map_x1) * 3;
			int img_data_idx21 = (map_y1 * dev_const_data[img_idx].width + map_x2) * 3;
			int img_data_idx22 = (map_y2 * dev_const_data[img_idx].width + map_x2) * 3;
			float total_weight = weight.total_weight[data_idx];
			
			for(int channel = 0; channel < 3; channel++)
			{
				dst[dst_data_idx + channel] += (unsigned char)((
					image[img_data_idx11 + channel] * dx2 * dy2 + 
					image[img_data_idx12 + channel] * dx2 * dy1 + 
					image[img_data_idx21 + channel] * dx1 * dy2 + 
					image[img_data_idx22 + channel] * dx1 * dy1
					) * total_weight);
			}
		}
	}
}

#define STREAM_NUM 2

int composeGPU(GPUImageData *images, unsigned char *dst)
{
	int pano_malloc_size = pano_size_.height * pano_size_.width * DST_IMAGE_CHANNEL * sizeof(unsigned char);
	CUDA_CHECK_CALL(hipMemset(dev_pano_, 0, pano_malloc_size), "hipMemset failed!\n", -2);
	
	for(int i = 0; i < image_num_; i++)
	{
		int img_size = const_data[i].height * const_data[i].width * 3 * sizeof(unsigned char);
		CUDA_CHECK_CALL(hipMemcpy(dev_imgs_[i].data, images[i].data, img_size, hipMemcpyHostToDevice), 
			"hipMemcpy images failed\n", -2);		//	2ms/f
		dim3 dimBlock(32, 16);
		dim3 dimGrid((const_data[i].warped_width + dimBlock.x - 1) / dimBlock.x, 
			(const_data[i].warped_height + dimBlock.y - 1) / dimBlock.y);
		compose<<<dimGrid, dimBlock>>>(dev_imgs_[i].data, dev_maps_[i], dev_weights_[i], dev_pano_, i, pano_size_);		//	4.1ms/f
	}
	
	CUDA_CHECK_CALL(hipDeviceSynchronize(), "hipDeviceSynchronize failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(dst, dev_pano_, pano_malloc_size, hipMemcpyDeviceToHost), 
		"hipMemcpy to dst failed\n", -2);			//	1.4ms/f
	return 0;
}

int freeGPU()
{
	for(int i = 0; i < image_num_; i++)
	{
		hipFree(dev_maps_[i].xmap);
		hipFree(dev_maps_[i].ymap);
		hipFree(dev_weights_[i].ec_weight);
		hipFree(dev_weights_[i].blend_weight);
		hipFree(dev_weights_[i].total_weight);
	}
	free(const_data);
	free(dev_imgs_);
	free(dev_maps_);
	free(dev_weights_);
	return 0;
}